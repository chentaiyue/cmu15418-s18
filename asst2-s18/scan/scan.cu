#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2. 
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__
void exclusive_scan_upsweep(int N, int twod, int twod1, int *output) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int base = (i + 1) * twod1 - 1;
    if (base < N) {
        output[base] += output[base - twod];
    }
}

__global__
void exclusive_scan_downsweep(int N, int twod, int twod1, int *output) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int base = (i + 1) * twod1 - 1;
    if (base < N) {
        int t = output[base - twod];
        output[base - twod] = output[base];
        output[base] += t;
    }
}

void exclusive_scan(int* device_start, int length, int* device_result)
{
    /* Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the input and output in device memory,
     * but this is host code -- you will need to declare one or more CUDA 
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the input and the output arrays are sized to accommodate the next
     * power of 2 larger than the input.
     */
    
    int rounded_length = nextPow2(length);
    if (rounded_length == 0) {
        return;
    }
    const int threadsPerBlock = 512;
    for (int twod = 1; twod < rounded_length; twod *= 2) {
        int twod1 = twod * 2;
        const int blocks = (rounded_length/twod1 + threadsPerBlock - 1) / threadsPerBlock;
        exclusive_scan_upsweep<<<blocks, threadsPerBlock>>>(rounded_length, twod, twod1, device_result);
    }
    hipMemset(device_result+rounded_length-1, 0, 1 * sizeof(int));
    for (int twod = rounded_length / 2; twod >= 1; twod /= 2) {
        int twod1 = twod * 2;
        const int blocks = (rounded_length/twod1 + threadsPerBlock - 1) / threadsPerBlock;
        exclusive_scan_downsweep<<<blocks, threadsPerBlock>>>(rounded_length, twod, twod1, device_result);
    }

}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input; 
    // We round the array sizes up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness. 
    // You may have an easier time in your implementation if you assume the 
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int length = end - inarray;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);
    hipMemcpy(device_input, inarray, length * sizeof(int), 
               hipMemcpyHostToDevice);

    // For convenience, both the input and output vectors on the device are
    // initialized to the input values. This means that you are free to simply
    // implement an in-place scan on the result vector if you wish.
    // If you do this, you will need to keep that fact in mind when calling
    // exclusive_scan from find_repeats.
    hipMemcpy(device_result, inarray, length * sizeof(int), 
               hipMemcpyHostToDevice);
    hipMemset(device_result+length, 0, (rounded_length - length) * sizeof(int));

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, end - inarray, device_result);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    
    hipMemcpy(resultarray, device_result, length * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), 
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}

__global__
void convert(int *a, int *b, int length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i + 1 < length && a[i] == a[i + 1]) {
        b[i] = 1;
    }
}

__global__
void collect(int *a, int *b, int *c, int length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i + 1 < length && a[i] == a[i + 1]) {
        c[b[i]] = i;
    }
}

int find_repeats(int *device_input, int *t, int length, int *device_output) {
    /* Finds all pairs of adjacent repeated elements in the list, storing the
     * indices of the first element of each pair (in order) into device_result.
     * Returns the number of pairs found.
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if 
     * it requires that. However, you must ensure that the results of
     * find_repeats are correct given the original length.
     */    
    int rounded_length = nextPow2(length);
    const int threadsPerBlock = 512;
    const int blocks = (length + threadsPerBlock - 1) / threadsPerBlock;
    convert<<<blocks, threadsPerBlock>>>(device_input, t, length);
    hipDeviceSynchronize();
    exclusive_scan(nullptr, length, t);
    collect<<<blocks, threadsPerBlock>>>(device_input, t, device_output, length);
    int num;
    hipMemcpy(&num, t + length - 1, 1 * sizeof(int), hipMemcpyDeviceToHost);
    return num;
}

/* Timing wrapper around find_repeats. You should not modify this function.
 */
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *t;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&t, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), 
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();
    int result = find_repeats(device_input, t, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(t);
    hipFree(device_output);

    return endTime - startTime;
}

void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
